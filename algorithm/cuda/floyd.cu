//
// Created by aminjon on 4/27/23.
//

#include <hip/hip_runtime.h>
#include <array>
#include <chrono>
#include <functional>
#include <iostream>
#include <random>

// #define PRINT_INITIAL_MATRIX
// #define PRINT_FINAL_MATRIX

constexpr int LB = 1;
constexpr int UB = 10;

std::random_device dev{};
std::mt19937 rnd(dev());
std::uniform_int_distribution<int> distribution(LB, UB);

constexpr int N = 2880;
constexpr int THREADS_PER_BLOCK = 16;// Each block have 16 * 16 = 256 threads
constexpr int BLOCKS_PER_SIDE = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
constexpr auto INF = std::numeric_limits<int>::max();

__global__ void work_gpu(int *adj, int k) {
  auto i = blockDim.y * blockIdx.y + threadIdx.y;
  auto j = blockDim.x * blockIdx.x + threadIdx.x;
  if (adj[i * N + k] == INF || adj[k * N + j] == INF) return;
  auto nDist = adj[i * N + k] + adj[k * N + j];
  if (i < N && j < N && (nDist < adj[i * N + j])) {
    adj[i * N + j] = nDist;
  }
}

int main() {
  hipError_t err = hipSuccess;

  int *hGraph = new int[N * N];
  [&]() {// init
    for (int i = 0; i < N; ++i) {
      for (int j = 0; j < N; ++j) {
        hGraph[i * N + j] = i == j
                                ? 0
                            : rnd() % 2 == 0
                                ? INF
                                : distribution(rnd);
      }
    }

#ifdef PRINT_INITIAL_MATRIX
    for (int i = 0; i < N; ++i) {
      for (int j = 0; j < N; ++j) {
        std::cout << (hGraph[i * N + j] == INF ? -1 : hGraph[i * N + j]) << (j + 1 < N ? ' ' : '\n');
      }
    }
#endif
  }();

  static_assert(N <= INF / N + INF % N);// overflow
  int *dGraph = nullptr;
  if ((err = hipMalloc(&dGraph, N * N * sizeof(int))) != hipSuccess) { std::cerr << "Failed while malloc dGraph_data: " << hipGetErrorString(err) << std::endl; std::exit(EXIT_FAILURE); }
  if ((err = hipMemcpy(dGraph, hGraph, N * N * sizeof(int), hipMemcpyHostToDevice)) != hipSuccess) { std::cerr << "Failed while memcpy hGraph: " << hipGetErrorString(err) << std::endl; std::exit(EXIT_FAILURE); }

  dim3 blocks(BLOCKS_PER_SIDE, BLOCKS_PER_SIDE, 1);
  dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);

  auto start = std::chrono::steady_clock::now();
  for (int k = 0; k < N; ++k) {
    work_gpu<<<blocks, threadsPerBlock>>>(dGraph, k);
  }
  hipDeviceSynchronize();
  auto dur = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start).count();

  if ((err = hipGetLastError()) != hipSuccess) { std::cerr << "Failed while launch kernel: " << hipGetErrorString(err) << std::endl; std::exit(EXIT_FAILURE); }

  if ((err = hipMemcpy(hGraph, dGraph, N * N * sizeof(int), hipMemcpyDeviceToHost)) != hipSuccess) { std::cerr << "Failed while memcpy dGraph: " << hipGetErrorString(err) << std::endl; std::exit(EXIT_FAILURE); }

#ifdef PRINT_FINAL_MATRIX
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      std::cout << (hGraph[i * N + j] == INF ? -1 : hGraph[i * N + j]) << (j + 1 < N ? ' ' : '\n');
    }
  }
#endif

  if ((err = hipFree(dGraph)) != hipSuccess) { std::cerr << "Failed while free dGraph: " << hipGetErrorString(err) << std::endl; std::exit(EXIT_FAILURE); }

  std::cout << "Duration: " << dur << std::endl;
}