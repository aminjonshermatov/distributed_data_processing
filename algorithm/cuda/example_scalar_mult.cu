//
// Created by aminjon on 4/27/23.
//

#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <array>
#include <iomanip>
#include <functional>

__global__ void vecAdd(const int *A, const int *B, int *C, int len) {
  auto i =blockDim.x * blockIdx.x + threadIdx.x;
  if (i < len) {
    C[i] = A[i] + B[i];
  }
}

constexpr int LB = 1;
constexpr int UB = 10;

std::random_device dev{};
std::mt19937 rnd(dev());
std::uniform_int_distribution<int> distribution(LB, UB);

constexpr int N = 50000;

int main() {
  hipError_t err = hipSuccess;

  std::array<int, N> hA{}, hB{}, hC{};
  [&](){ // init
    std::generate(hA.begin(), hA.end(), std::bind(distribution, std::ref(rnd)));
    std::generate(hB.begin(), hB.end(), std::bind(distribution, std::ref(rnd)));

    std::cout << "N: " << N << std::endl;
    for (int i = 0; i < N; ++i) {
      std::cout << std::setw(2) << std::setfill('0') << hA[i] << (i + 1 < N ? ' ' : '\n');
    }
    for (int i = 0; i < N; ++i) {
      std::cout << std::setw(2) << std::setfill('0') << hB[i] << (i + 1 < N ? ' ' : '\n');
    }
  }();

  int *dA = nullptr, *dB = nullptr, *dC = nullptr;
  if ((err = hipMalloc(&dA, N * sizeof(int))) != hipSuccess) { std::cerr << "Failed while malloc A: " << hipGetErrorString(err) << std::endl; }
  if ((err = hipMalloc(&dB, N * sizeof(int))) != hipSuccess) { std::cerr << "Failed while malloc B: " << hipGetErrorString(err) << std::endl; }
  if ((err = hipMalloc(&dC, N * sizeof(int))) != hipSuccess) { std::cerr << "Failed while malloc C: " << hipGetErrorString(err) << std::endl; }

  if ((err = hipMemcpy(dA, hA.data(), N * sizeof(int), hipMemcpyHostToDevice)) != hipSuccess) { std::cerr << "Failed while memcpy A: " << hipGetErrorString(err) << std::endl; }
  if ((err = hipMemcpy(dB, hB.data(), N * sizeof(int), hipMemcpyHostToDevice)) != hipSuccess) { std::cerr << "Failed while memcpy B: " << hipGetErrorString(err) << std::endl; }

  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  std::cout << "threadsPerBlock: " << threadsPerBlock << " blocksPerGrid: " << blocksPerGrid << std::endl;

  vecAdd<<<blocksPerGrid, threadsPerBlock>>>(dA, dB, dC, N);
  if ((err = hipGetLastError()) != hipSuccess) { std::cerr << "Failed while launch kernel: " << hipGetErrorString(err) << std::endl; }

  if ((err = hipMemcpy(hC.data(), dC, N * sizeof(int), hipMemcpyDeviceToHost)) != hipSuccess) { std::cerr << "Failed while memcpy C: " << hipGetErrorString(err) << std::endl; }

  for (int i = 0; i < N; ++i) {
    std::cout << std::setw(2) << std::setfill('0') << hC[i] << (i + 1 < N ? ' ' : '\n');
  }

  if ((err = hipFree(dA)) != hipSuccess) { std::cerr << "Failed while free A: " << hipGetErrorString(err) << std::endl; }
  if ((err = hipFree(dB)) != hipSuccess) { std::cerr << "Failed while free B: " << hipGetErrorString(err) << std::endl; }
  if ((err = hipFree(dC)) != hipSuccess) { std::cerr << "Failed while free C: " << hipGetErrorString(err) << std::endl; }

}